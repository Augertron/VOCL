#include "hip/hip_runtime.h"
#include "cudacommon.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

#include <cassert>
#include <iostream>
#include <vector>

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Scan.h"
#include "scan_kernel.h"

using namespace std;

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op)
{
    op.addOption("iterations", OPT_INT, "256", "specify scan iterations");
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the scan (parallel prefix sum) benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void
RunBenchmark(ResultDatabase &resultDB, OptionParser &op)
{
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    cout << "Running single precision test" << endl;
    RunTest<float, float4>("Scan", resultDB, op);

    // Test to see if this device supports double precision
    if ((deviceProp.major == 1 && deviceProp.minor >= 3) ||
               (deviceProp.major >= 2))
    {
        cout << "Running double precision test" << endl;
        RunTest<double, double4>("Scan-DP", resultDB, op);
    } else {
        cout << "Skipping double precision test" << endl;
        char atts[1024] = "DP_Not_Supported";
        // resultDB requires neg entry for every possible result
        int passes = op.getOptionInt("passes");
        for (int k = 0; k < passes; k++) {
            resultDB.AddResult("Scan-DP" , atts, "GB/s", FLT_MAX);
            resultDB.AddResult("Scan-DP_PCIe" , atts, "GB/s", FLT_MAX);
            resultDB.AddResult("Scan-DP_Parity" , atts, "GB/s", FLT_MAX);
        }
    }
}

template <class T, class vecT>
void RunTest(string testName, ResultDatabase &resultDB, OptionParser &op)
{
    int probSizes[4] = { 1, 8, 32, 64 };

    int size = probSizes[op.getOptionInt("size")-1];
    // Convert to MB
    size = (size * 1024 * 1024) / sizeof(T);
    // create input data on CPU
    unsigned int bytes = size * sizeof(T);

    // Allocate Host Memory
    T* h_idata;
    T* reference;
    T* h_odata;
    CUDA_SAFE_CALL(hipHostMalloc((void**) &h_idata,   bytes));
    CUDA_SAFE_CALL(hipHostMalloc((void**) &reference, bytes));
    CUDA_SAFE_CALL(hipHostMalloc((void**) &h_odata,   bytes));

    // Initialize host memory
    cout << "Initializing host memory." << endl;
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = i % 3; // Fill with some pattern
        h_odata[i] = i % 3;
    }

    // allocate device memory
    T* d_idata, *d_odata;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_idata, bytes));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_odata, bytes));

    // Copy data to GPU
    cout << "Copying data to device." << endl;
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    CUDA_SAFE_CALL(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    // Get elapsed time
    float transferTime = 0.0f;
    hipEventElapsedTime(&transferTime, start, stop);
    transferTime *= 1.e-3;

    //Allocate space for block sums
    int numLevelsAllocated = 0;
    unsigned int maxNumElements = size;
    unsigned int numElts = size;
    int level = 0;
    do
    {
        unsigned int numBlocks = max(1, (int) ceil((float) numElts /
                (2.f * BLOCK_SIZE)));
        if (numBlocks > 1)
        {
            level++;
        }
        numElts = numBlocks;
    }
    while (numElts > 1);

    T** scanBlockSums = (T**) malloc((level + 1) * sizeof(T*));
    assert(scanBlockSums != NULL);
    numLevelsAllocated = level + 1;
    numElts = maxNumElements;
    level = 0;

    do
    {
        unsigned int numBlocks = max(1, (int) ceil((float) numElts / (4.f
                * BLOCK_SIZE)));
        if (numBlocks > 1)
        {
            //Malloc GPU Mem for block sums
            CUDA_SAFE_CALL(hipMalloc((void**)&(scanBlockSums[level]),
                    numBlocks*sizeof(T)));
            level++;
        }
        numElts = numBlocks;
    }
    while (numElts > 1);

    CUDA_SAFE_CALL(hipMalloc((void**)&(scanBlockSums[level]),
            sizeof(T)));

    int passes = op.getOptionInt("passes");
    int iters = op.getOptionInt("iterations");

    cout << "Running benchmark with size " << size << endl;
    for (int k = 0; k < passes; k++)
    {
        float totalScanTime = 0.0f;
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
        for (int j = 0; j < iters; j++)
        {
            scanArrayRecursive<T, vecT>
                (d_odata, d_idata, size, 0, scanBlockSums);
        }
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        hipEventElapsedTime(&totalScanTime, start, stop);

        float oTransferTime = 0.0f;
        CUDA_SAFE_CALL(hipEventRecord(start, 0));
        CUDA_SAFE_CALL(hipMemcpy(h_odata, d_odata, bytes,
                hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipEventRecord(stop, 0));
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        hipEventElapsedTime(&oTransferTime, start, stop);

        // Only add output transfer time once
        if (k == 0)
        {
            transferTime += oTransferTime;
        }

        // If results aren't correct, don't report perf numbers
        if (! scanCPU<T>(h_idata, reference, h_odata, size))
        {
            return;
        }

        char atts[1024];
        double avgTime = (totalScanTime / (double) iters);
        avgTime *= 1.e-3;
        sprintf(atts, "%d items", size);
        double gb = (double)(size * sizeof(T)) / (1000. * 1000. * 1000.);
        resultDB.AddResult(testName, atts, "GB/s", gb / avgTime);
        resultDB.AddResult(testName+"_PCIe", atts, "GB/s",
                gb / (avgTime + transferTime));
        resultDB.AddResult(testName+"_Parity", atts, "N",
                transferTime / avgTime);

    }
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUDA_SAFE_CALL(hipHostFree(h_idata));
    CUDA_SAFE_CALL(hipHostFree(h_odata));
    CUDA_SAFE_CALL(hipHostFree(reference));
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    for (int i = 0; i < numLevelsAllocated; i++)
    {
        CUDA_SAFE_CALL(hipFree(scanBlockSums[i]));
    }
    free(scanBlockSums);
}

// ****************************************************************************
// Function: scanArrayRecursive
//
// Purpose:
//   Workhorse for the scan benchmark, this function recursively scans
//   arbitrary sized arrays, including those which are of a non power
//   of two length, or not evenly divisible by block size
//
// Arguments:
//     outArray: pointer to output memory on the device
//     inArray:  pointer to input memory on the device
//     numElements: the number of elements to scan
//     level: the current level of recursion, starting at 0
//     blockSums: pointer to device memory to store intermediate sums
//
// Returns:
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
template <class T, class vecT>
void scanArrayRecursive(T* outArray, T* inArray, int numElements,
        int level, T** blockSums)
{
    // Kernels handle 8 elems per thread
    unsigned int numBlocks = max(1,
    		(unsigned int)ceil((float)numElements/(4.f * BLOCK_SIZE)));
    unsigned int sharedEltsPerBlock = BLOCK_SIZE * 2;
    unsigned int sharedMemSize = sizeof(T) * sharedEltsPerBlock;

    bool fullBlock = (numElements == numBlocks * 4 * BLOCK_SIZE);

    dim3 grid(numBlocks, 1, 1);
    dim3 threads(BLOCK_SIZE, 1, 1);

    // execute the scan
    if (numBlocks > 1)
    {
        scan<T, vecT><<<grid, threads, sharedMemSize>>>
            (outArray, inArray, blockSums[level], numElements, fullBlock, true);
    } else
    {
        scan<T, vecT><<<grid, threads, sharedMemSize>>>
           (outArray, inArray, blockSums[level], numElements, fullBlock, false);
    }
    if (numBlocks > 1)
    {
        scanArrayRecursive<T, vecT>(blockSums[level], blockSums[level],
                numBlocks, level + 1, blockSums);
        vectorAddUniform4<T><<< grid, threads >>>
                (outArray, blockSums[level], numElements);
    }
}

// ****************************************************************************
// Function: scanCPU
//
// Purpose:
//   Simple cpu scan routine to verify device results
//
// Arguments:
//   data : the input data
//   reference : space for the cpu solution
//   dev_result : result from the device
//   size : number of elements
//
// Returns:  nothing, prints relevant info to stdout
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
template <class T>
bool scanCPU(T *data, T* reference, T* dev_result, const size_t size)
{
    reference[0] = 0;
    bool passed = true;

    for (unsigned int i = 1; i < size; ++i)
    {
        reference[i] = data[i - 1] + reference[i - 1];
    }
    for (unsigned int i = 0; i < size; ++i)
    {
        if (reference[i] != dev_result[i])
        {
#ifdef VERBOSE_OUTPUT
            cout << "Mismatch at i: " << i << " ref: " << reference[i]
                 << " dev: " << dev_result[i] << endl;
#endif
            passed = false;
        }
    }
    cout << "Test ";
    if (passed)
        cout << "Passed" << endl;
    else
        cout << "---FAILED---" << endl;
    return passed;
}
